﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#define VALID_BIT 31
#define DIRTY_BIT 30
#define GET_BIT(b,t) ((t & (0x1<<b))>>b)
#define SET_BIT(b,t) t = t | (0x1<<b)
#define UNSET_BIT(b,t) t = t & ~(0x1<<b)
#define GET_PID(vm,index) ((vm->invert_page_table[index*2]>>13) & 0x3)
#define GET_ADDR(vm,index) (vm->invert_page_table[index*2] & 0x1FFF)
#define GET_NEXT(vm,index) (vm->invert_page_table[index*2+1] & 0x3FF)
#define GET_PREV(vm,index) ((vm->invert_page_table[index*2+1]>>11) & 0x3FF)
#define IS_EMPTY(vm) GET_BIT(22,vm->invert_page_table[5])
#define SET_EMPTY(vm) SET_BIT(22,vm->invert_page_table[5])
#define UNSET_EMPTY(vm) UNSET_BIT(22,vm->invert_page_table[5])
#define GET_HEAD_NODE(vm) ((vm->invert_page_table[1]>>22) & 0x3FF)
#define GET_TAIL_NODE(vm) ((vm->invert_page_table[3]>>22) & 0x3FF)
#define SET_HEAD_NODE(vm,head) vm->invert_page_table[1] = ((vm->invert_page_table[1] & ~(0x3FF<<22))|((head&0x3FF)<<22))
#define SET_TAIL_NODE(vm,tail) vm->invert_page_table[3] = ((vm->invert_page_table[3] & ~(0x3FF<<22))|((tail&0x3FF)<<22))
#define IS_DIRTY(vm,index) GET_BIT(VALID_BIT,vm->invert_page_table[index*2])
#define IS_INVALID(vm,index) GET_BIT(DIRTY_BIT,vm->invert_page_table[index*2])
#define SET_DIRTY(vm,index) SET_BIT(VALID_BIT,vm->invert_page_table[index*2])
#define SET_INVALID(vm,index) SET_BIT(DIRTY_BIT,vm->invert_page_table[index*2])
#define SET_PID (vm,index,pid) vm->invert_page_table[index*2] = (vm->invert_page_table[index*2] & (~(0x3<<13)) | ((pid & 0x3) << 13))
#define UNSET_DIRTY(vm,index) UNSET_BIT(VALID_BIT,vm->invert_page_table[index*2])
#define UNSET_INVALID(vm,index) UNSET_BIT(DIRTY_BIT,vm->invert_page_table[index*2])
#define SET_ADDR(vm,index,addr) vm->invert_page_table[index*2] = (vm->invert_page_table[index*2]& ~(0x1FFF))|(addr & 0x1FFF)
#define SET_NEXT(vm,index,next_index) vm->invert_page_table[index*2+1] =\
(vm->invert_page_table[index*2+1] & ~(0x3FF))|(next_index & 0x3FF); UNSET_NEXT_NULL(vm,index);
#define SET_PREV(vm,index,prev_index) vm->invert_page_table[index*2+1] =\
(vm->invert_page_table[index*2+1] & ~(0x3FF<<11))| ((prev_index & 0x3FF)<<11); UNSET_PREV_NULL(vm,index);
#define SET_NEXT_NULL(vm,index) SET_BIT(10,vm->invert_page_table[index*2+1])
#define SET_PREV_NULL(vm,index) SET_BIT(21,vm->invert_page_table[index*2+1])
#define UNSET_NEXT_NULL(vm,index) UNSET_BIT(10,vm->invert_page_table[index*2+1])
#define UNSET_PREV_NULL(vm,index) UNSET_BIT(21,vm->invert_page_table[index*2+1])
#define NEXT_IS_NULL(vm,index) GET_BIT(10,vm->invert_page_table[index*2+1])
#define PREV_IS_NULL(vm,index) GET_BIT(21,vm->invert_page_table[index*2+1])
#define INIT(vm,index) vm->invert_page_table[index*2] = 0; vm->invert_page_table[index*2+1] = 0
#define INIT_DISK_MAP(vm,index) vm->invert_page_table[vm->PAGE_ENTRIES*2+index/2] = \
(vm->invert_page_table[vm->PAGE_ENTRIES*2+index/2] & ~(0xFFFF<<((index%2)*16))) | ((((1<<15))) << ((index%2)*16))
#define GET_DISK_IMAP(vm,index) ((vm->invert_page_table[vm->PAGE_ENTRIES*2+index/2]) & (0xFFFF<<((index%2)*16))) >> ((index%2)*16)
#define GET_VM_FROM_DISK(vm,index) (GET_DISK_IMAP(vm,index) & 0x1FFF)
#define GET_PID_FROM_DISK(vm,index) ((GET_DISK_IMAP(vm,index) & 0x3<<13)>>13)
#define DISK_IS_INVALID(vm,index) GET_BIT(15,GET_DISK_IMAP(vm,index))
#define SET_DISK_INVALID(vm,index) SET_BIT(15+(index%2)*16,vm->invert_page_table[vm->PAGE_ENTRIES*2+index/2])
#define UNSET_DISK_INVALID(vm,index) UNSET_BIT(15+(index%2)*16,vm->invert_page_table[vm->PAGE_ENTRIES*2+index/2])
#define CONSTRUCT_DISK_IMAP(pid,vm_addr) ((((vm_addr & 0x1FFF) | ((pid & 0x3) <<13))) & ~(1<<15))
#define SET_DISK_TO_VM(vm,index,pid,vm_addr) vm->invert_page_table[vm->PAGE_ENTRIES*2+index/2] = \
(vm->invert_page_table[vm->PAGE_ENTRIES*2+index/2] & ~(0xFFFF<<((index%2)*16)) | (CONSTRUCT_DISK_IMAP(pid,vm_addr) << ((index%2)*16)))
#define SET_COUNT(vm,count) (vm->invert_page_table[(vm->PAGE_ENTRIES-1)*2] = ((vm->invert_page_table[(vm->PAGE_ENTRIES-1)*2] & ~(0x7FF<<15)) | (count<<15)))
#define GET_COUNT(vm) ((vm->invert_page_table[(vm->PAGE_ENTRIES-1)*2] & (0x7FF<<15)) >> 15) //11 bit count
#define HASH(key,i) (key%1021+i)%1024
#define HASH_DISK(key,i) (key%4099+i)%4096


__device__ void init_invert_page_table(VirtualMemory *vm) {
    for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
        INIT(vm,i);
        SET_INVALID(vm,i);
        SET_PREV_NULL(vm,i);
        SET_NEXT_NULL(vm,i);
    }

    for (int i=0; i < (vm->STORAGE_SIZE/vm->PAGESIZE);i++)
        INIT_DISK_MAP(vm,i);
    SET_COUNT(vm,0);
    SET_EMPTY(vm);

}
/** Usage:
  vm_init(&vm, data, storage,
          pt, &pagefault_num,
          PAGE_SIZE, INVERT_PAGE_TABLE_SIZE,
          PHYSICAL_MEM_SIZE, STORAGE_SIZE,
          PHYSICAL_MEM_SIZE / PAGE_SIZE);
**/
__device__ void vm_init(VirtualMemory *vm, uchar *buffer, uchar *storage,
                        u32 *invert_page_table, int *pagefault_num_ptr,
                        int PAGESIZE, int INVERT_PAGE_TABLE_SIZE,
                        int PHYSICAL_MEM_SIZE, int STORAGE_SIZE,
                        int PAGE_ENTRIES) {
  // init variables
  vm->buffer = buffer;
  vm->storage = storage;
  vm->invert_page_table = invert_page_table;
  vm->pagefault_num_ptr = pagefault_num_ptr;

  // init constants
  vm->PAGESIZE = PAGESIZE;
  vm->INVERT_PAGE_TABLE_SIZE = INVERT_PAGE_TABLE_SIZE;
  vm->PHYSICAL_MEM_SIZE = PHYSICAL_MEM_SIZE;
  vm->STORAGE_SIZE = STORAGE_SIZE;
  vm->PAGE_ENTRIES = PAGE_ENTRIES;

  // before first vm_write or vm_read
  init_invert_page_table(vm);
}

__device__ void mark_use(VirtualMemory *vm, u32 mem_entry) {
    /** MOVE MEM_ENTRY TO HEAD IN LINKED LIST **/
    if(IS_EMPTY(vm)) {
        SET_HEAD_NODE(vm,mem_entry);
        SET_TAIL_NODE(vm,mem_entry);
        UNSET_EMPTY(vm);
    }
    /** DELETE NODE FROM LIST **/
    if(GET_HEAD_NODE(vm)!=mem_entry){
        
        if(NEXT_IS_NULL(vm,mem_entry)==0){
        if(PREV_IS_NULL(vm,mem_entry)==0){
            SET_PREV(vm,GET_NEXT(vm,mem_entry),GET_PREV(vm,mem_entry));
        }
        else{
            SET_PREV_NULL(vm,GET_NEXT(vm,mem_entry));
        }
    }
    
    if(PREV_IS_NULL(vm,mem_entry)==0){
        if(NEXT_IS_NULL(vm,mem_entry)==0) {
            
            SET_NEXT(vm,GET_PREV(vm,mem_entry),GET_NEXT(vm,mem_entry));
        }
        else {
            SET_NEXT_NULL(vm,GET_PREV(vm,mem_entry));
        }
            
    }
    
    u32 old_head = GET_HEAD_NODE(vm);
    SET_NEXT(vm,mem_entry,old_head);
    SET_PREV(vm,old_head,mem_entry);
    SET_HEAD_NODE(vm,mem_entry);
        
        
    }
    
    
}

__device__ void swap_in(VirtualMemory *vm,u32 mem_entry, u32 disk_entry) {
    for(u32 i=0;i<vm->PAGESIZE;i++)
       vm->buffer[(mem_entry<<5|(i&0x1F))] = vm->storage[(disk_entry<<5|(i&0x1F))];
    UNSET_DIRTY(vm,mem_entry); // fresh entry certainly not dirty
}

__device__ void swap_out(VirtualMemory *vm,u32 mem_entry, u32 disk_entry, u32 vm_addr) {
    UNSET_DISK_INVALID(vm,disk_entry);
    SET_DISK_TO_VM(vm,disk_entry,0,vm_addr);
    for(u32 i=0;i<vm->PAGESIZE;i++)
      vm->storage[(disk_entry<<5|(i&0x1F))] = vm->buffer[(mem_entry<<5|(i&0x1F))];
}

__device__ u32 extract_lru(VirtualMemory *vm) {
    u32 target = GET_TAIL_NODE(vm);
    if(PREV_IS_NULL(vm,target)==0) {
      /** target not head, therefore has prev **/
      u32 new_tail = GET_PREV(vm,target);
      SET_TAIL_NODE(vm,new_tail);
    }
    else SET_EMPTY(vm);
    return target;
}


__device__ u32 evict_lru(VirtualMemory *vm) {
    u32 target = extract_lru(vm);
    u32 disk_entry;
    u32 vm_page_addr = GET_ADDR(vm,target);
    if(IS_DIRTY(vm,target)){
      for(u32 i=0; i < (vm->STORAGE_SIZE/vm->PAGESIZE); i++) {
      disk_entry = HASH_DISK(vm_page_addr,i);
      if(DISK_IS_INVALID(vm,disk_entry)) break;
    }
    //printf("SWAPPED OUT MEM #%d to DISK #%d",target,disk_entry);
    swap_out(vm,target,disk_entry,vm_page_addr);
    }
    
    return target;
}

__device__ uchar vm_read(VirtualMemory *vm, u32 addr) {
  /* Complate vm_read function to read single element from data buffer */

  u32 vm_page_addr = (addr >> 5);
  u32 vm_page_offset = addr & 0x1F;
  if(GET_COUNT(vm) < vm->PAGE_ENTRIES) {
    /** non-full case, use hashing **/
    /** no need to consider replacement **/
    u32 mem_entry; // this can also mark the first empty slot appearing
    for(u32 i=0; i < vm->PAGE_ENTRIES; i++) {
      mem_entry = HASH(vm_page_addr,i);
      if(IS_INVALID(vm,mem_entry)) break;
      if(GET_ADDR(vm,mem_entry) == vm_page_addr) {
        mark_use(vm,mem_entry);
        return vm->buffer[((mem_entry<<5) | vm_page_offset)];
      }

    }
    /** unable to find in mem, page fault check disk **/
    (*(vm->pagefault_num_ptr))++;
    for(u32 i=0; i < (vm->STORAGE_SIZE/vm->PAGESIZE); i++) {
      u32 disk_entry = HASH_DISK(vm_page_addr,i);
      if(DISK_IS_INVALID(vm,disk_entry)) break; //cannot found in disk either, error
      if(GET_VM_FROM_DISK(vm,disk_entry)==vm_page_addr) {
        /* page allocation routine */
        UNSET_INVALID(vm,mem_entry);
        SET_ADDR(vm,mem_entry,vm_page_addr);
        SET_COUNT(vm,GET_COUNT(vm)+1);
        /* swap_in and mark use */
        swap_in(vm,mem_entry,disk_entry);
        mark_use(vm,mem_entry);
        //return;
        return vm->buffer[((mem_entry<<5) | vm_page_offset)];
      }
    }

  }
  else {
    /** full case, linear tranverse **/
    for(u32 mem_entry=0; mem_entry < vm->PAGE_ENTRIES; mem_entry++) {
      if(IS_INVALID(vm,mem_entry)==0 && GET_ADDR(vm,mem_entry) == vm_page_addr){
        mark_use(vm,mem_entry);
        //printf("Page found: #%d\n",mem_entry);
        return vm->buffer[((mem_entry<<5) | vm_page_offset)];
      }
    }
    
    /** unable to find in mem, page fault and check disk **/
    (*(vm->pagefault_num_ptr))++;
    for(u32 i=0; i < (vm->STORAGE_SIZE/vm->PAGESIZE); i++) {
      u32 disk_entry = HASH_DISK(vm_page_addr,i);
      if(DISK_IS_INVALID(vm,disk_entry)) break;
      //printf("DISK_VM: #%d MY_VM: #%d\n",GET_VM_FROM_DISK(vm,disk_entry),vm_page_addr);
      if(GET_VM_FROM_DISK(vm,disk_entry)==vm_page_addr) {
        //printf("Found on disk page: #%d\n",disk_entry);
        u32 destination = evict_lru(vm);
        /* page allocation routine */
        UNSET_INVALID(vm,destination);
        SET_ADDR(vm,destination,vm_page_addr);
        // SET_COUNT(vm,GET_COUNT(vm)+1);
        // COUNT ALREADY MAX

        /* swap in and mark use */
        swap_in(vm,destination,disk_entry);
        mark_use(vm,destination);
        //return;
        return vm->buffer[((destination<<5) | vm_page_offset)];
      }
    }
  
  

  }
 return 1;
}


__device__ void vm_write(VirtualMemory *vm, u32 addr, uchar value) {
  /* Complete vm_write function to write value into data buffer */
  u32 vm_page_addr = (addr >> 5);
  u32 vm_page_offset = addr & 0x1F;
  if(GET_COUNT(vm) < vm->PAGE_ENTRIES) {
    /** non-full case, use hashing **/
    /** no need to consider replacement **/
    u32 mem_entry; // this can also mark the first empty slot appearing
    for(u32 i=0; i < vm->PAGE_ENTRIES; i++) {
      mem_entry = HASH(vm_page_addr,i);
      if(IS_INVALID(vm,mem_entry)) break;
      if(GET_ADDR(vm,mem_entry) == vm_page_addr) {
        mark_use(vm,mem_entry);
        vm->buffer[((mem_entry<<5) | vm_page_offset)] = value;
        SET_DIRTY(vm,mem_entry);
        return;
      }
    }
    //printf("Direct write to page#%d\n",mem_entry);
    /** no record in mem, write directly to mem_entry **/
    /** page allocation routine, page fault as well **/
    (*(vm->pagefault_num_ptr))++;
    UNSET_INVALID(vm,mem_entry);
    SET_COUNT(vm,GET_COUNT(vm)+1);
    SET_DIRTY(vm,mem_entry); // dirty at born
    /** record new vm mapping **/
    SET_ADDR(vm,mem_entry,vm_page_addr);
    vm->buffer[((mem_entry<<5) | vm_page_offset)] = value;
    mark_use(vm,mem_entry);
  }
  else {
    /** full case, linear tranverse **/
    for(u32 mem_entry=0; mem_entry < vm->PAGE_ENTRIES; mem_entry++) {
      if(GET_ADDR(vm,mem_entry) == vm_page_addr){
        SET_DIRTY(vm,mem_entry);
        mark_use(vm,mem_entry);
        vm->buffer[((mem_entry<<5) | vm_page_offset)] = value;
        return;
      }
    }
    
    /** unable to find in mem, page fault, evict a victim and place it there **/
    (*(vm->pagefault_num_ptr))++;
    u32 destination = evict_lru(vm);
    //printf("Evict vitcim: #%d\n",destination);
    UNSET_INVALID(vm,destination);
    /** record new vm mapping **/
    SET_ADDR(vm,destination,vm_page_addr);
    SET_DIRTY(vm,destination);
    mark_use(vm,destination);
    vm->buffer[((destination<<5) | vm_page_offset)] = value;
    return;

 }
}


__device__ void vm_snapshot(VirtualMemory *vm, uchar *results, int offset,
                            int input_size) {
  /* Complete snapshot function togther with vm_read to load elements from data
   * to result buffer */
   for(u32 i=0;i<input_size;i++)
      results[i] = vm_read(vm,offset+i);
}