﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ void user_program(VirtualMemory *vm, uchar *input, uchar *results,
                             int input_size, Lock* lock_ptr, u32 pid) {
  for (int i = 0; i < input_size; i++)
    
    vm_write(vm, i, input[i],lock_ptr,pid);

  for (int i = input_size - 1; i >= input_size - 32769; i--)
    int value = vm_read(vm, i,lock_ptr,pid);

  vm_snapshot(vm, results, 0, input_size,lock_ptr,pid);
}
